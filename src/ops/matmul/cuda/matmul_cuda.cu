#include "hip/hip_runtime.h"
#include "../../../devices/cuda/cuda_handle.h"
#include "../../utils.h"
#include "../blas.h"
#include "matmul_cuda.h"
#include <hipblas.h>
#include <hip/hip_fp16.h>

void matmul_cuda_f16(MatmulCudaDescriptor_t desc, void *c, float beta, void const *a, void const *b, float alpha, void *stream) {
    auto info = desc->info;

    if (info.is_transed) {
        std::swap(a, b);
    }

    auto alpha_f16 = __float2half(alpha);
    auto beta_f16 = __float2half(beta);

    auto op_a = info.a_matrix.row_stride == 1 ? HIPBLAS_OP_N : HIPBLAS_OP_T;
    auto op_b = info.b_matrix.row_stride == 1 ? HIPBLAS_OP_N : HIPBLAS_OP_T;

    use_cublas(desc->cublas_handles_t, desc->device_id, (hipStream_t) stream,
               [&](hipblasHandle_t handle) { hipblasGemmStridedBatchedEx(
                                                handle,
                                                op_a,
                                                op_b,
                                                info.m,
                                                info.n,
                                                info.k,
                                                &alpha_f16,
                                                a,
                                                HIP_R_16F,
                                                info.a_matrix.ld(),
                                                info.a_matrix.stride,
                                                b,
                                                HIP_R_16F,
                                                info.b_matrix.ld(),
                                                info.b_matrix.stride,
                                                &beta_f16,
                                                c,
                                                HIP_R_16F,
                                                info.c_matrix.ld(),
                                                info.c_matrix.stride,
                                                info.batch,
                                                HIPBLAS_COMPUTE_16F,
                                                CUBLAS_GEMM_DEFAULT_TENSOR_OP); });
}
