#include "hip/hip_runtime.h"
#include "../../../devices/cuda/common_cuda.h"
#include "../../utils.h"
#include "layer_norm.cuh"
#include <hipcub/hipcub.hpp>

template <typename T, int BLOCK_DIM>
__launch_bounds__(BLOCK_DIM)
    __global__ void blockLayernormKernel(T const *input, T const *scale, T const *bias, T *output, float eps, int behindsize)
{
    // 假设input= [A, B, C, D], axis = 2, frontsize = AB = blockDim.x, behindsize = CD
    // 全局索引index = i(BCD) + j (CD) + k(D) + s
    // blockIdx.x = i(B) + j;默认behindsize >= BLOCK_DIM
    // scale,bias长度为behindsize,形状为[C,D]
    int tid = blockIdx.x * behindsize;
    float muPartial = 0.0f;
    for (int id = threadIdx.x; id < behindsize; id += BLOCK_DIM)
    {
        muPartial += static_cast<float>(input[tid + id]); // half很多操作不支持，运算过程使用float数据
    }
    typedef hipcub::BlockReduce<float, BLOCK_DIM> BlockReduce;
    __shared__ typename BlockReduce::TempStorage temp_storage;
    __shared__ float mu;
    float muBlock = BlockReduce(temp_storage).Reduce(muPartial, hipcub::Sum());
    if (threadIdx.x == 0)
    {
        mu = muBlock * __fdividef(1.0F, behindsize);
    } // threadIdx.x = 0对应的是全局sum
    __syncthreads();
    float sigma2Partial = 0.0f;
    for (int id = threadIdx.x; id < behindsize; id += BLOCK_DIM)
    {
        sigma2Partial += (static_cast<float>(input[tid + id]) - mu) * (static_cast<float>(input[tid + id]) - mu);
    }
    __shared__ float sigma2;
    float sigma2Block = BlockReduce(temp_storage).Reduce(sigma2Partial, hipcub::Sum());
    if (threadIdx.x == 0)
    {
        float sigmaTmp = sqrt(sigma2Block * __fdividef(1.0F, behindsize) + eps);
        sigma2 = __fdividef(1.0F, sigmaTmp);
    }
    __syncthreads();
    for (int id = threadIdx.x; id < behindsize; id += BLOCK_DIM)
    {
        output[tid + id] = static_cast<T>(static_cast<float>(scale[id]) * (static_cast<float>(input[tid + id]) - mu) * sigma2 + static_cast<float>(bias[id]));
    }
}
template <typename T>
struct SumOp
{
    __device__ __forceinline__ T operator()(const T &a, const T &b) const
    {
        return a + b;
    }
};

template <template <typename> class ReductionOp, typename T,
          int thread_group_width>
__inline__ __device__ T WarpAllReduce(T val)
{
    for (int mask = thread_group_width / 2; mask > 0; mask /= 2)
    {
        val = ReductionOp<T>()(val, __shfl_xor_sync(0xffffffff, val, mask));
    }
    return val;
}
template <typename T, int BLOCK_DIM_x, int BLOCK_DIM_y>
__global__ void warpLayernormKernel(T const *input, T const *scale, T const *bias, T *output, float eps, int behindsize)
{
    // 默认behindsize < 1024
    int otherIdx = blockIdx.x * blockDim.y + threadIdx.y;
    int tid = otherIdx * behindsize;
    float muPartial = 0.0f;
    for (int id = threadIdx.x; id < behindsize; id += BLOCK_DIM_x)
    {
        muPartial += static_cast<float>(input[tid + id]);
    }
    muPartial = WarpAllReduce<SumOp, float, BLOCK_DIM_x>(muPartial);
    __shared__ float mu[BLOCK_DIM_y];

    if (threadIdx.x == 0)
    {
        mu[threadIdx.y] = muPartial * __fdividef(1.0F, behindsize);
    } // threadIdx.x = 0对应的是全局sum
    __syncthreads();
    float sigma2Partial = 0.0f;
    for (int id = threadIdx.x; id < behindsize; id += BLOCK_DIM_x)
    {
        sigma2Partial += (static_cast<float>(input[tid + id]) - mu[threadIdx.y]) * (static_cast<float>(input[tid + id]) - mu[threadIdx.y]);
    }
    sigma2Partial = WarpAllReduce<SumOp, float, BLOCK_DIM_x>(sigma2Partial);
    __shared__ float sigma2[BLOCK_DIM_y];

    if (threadIdx.x == 0)
    {
        float sigmaTmp = sqrt(sigma2Partial * __fdividef(1.0F, behindsize) + eps);
        sigma2[threadIdx.y] = __fdividef(1.0F, sigmaTmp);
    }
    __syncthreads();
    for (int id = threadIdx.x; id < behindsize; id += BLOCK_DIM_x)
    {
        output[tid + id] = static_cast<T>(static_cast<float>(scale[id]) * (static_cast<float>(input[tid + id]) - mu[threadIdx.y]) * sigma2[threadIdx.y] + static_cast<float>(bias[id]));
    }
}

template<typename T>
void layer_norm_nv_gpu(LayerNormCudaDescriptor_t desc, void const *input, void const *scale, void const *bias, void *output) {
    int size = desc->size;
    int behindsize = desc->behindsize;
    int num_blocks = size / behindsize;
    if (behindsize >= 1024)
    {
        int BLOCK_DIM = 1024;
        blockLayernormKernel<T, 1024>
            <<<num_blocks, BLOCK_DIM, 0, (hipStream_t) stream>>>((T *)input, (T *)scale, (T *)bias, (T *)output, eps, behindsize);
    }
    else if (behindsize > 31)
    {
        int BLOCK_DIM_x = 32;
        int BLOCK_DIM_y = 32;
        int num_block_x = (num_blocks + BLOCK_DIM_y - 1) / BLOCK_DIM_y;
        dim3 block_dim(BLOCK_DIM_x, BLOCK_DIM_y, 1);
        dim3 grid_dim(num_block_x, 1, 1);

        warpLayernormKernel<T, 32, 32>
            <<<grid_dim, block_dim, 0, (hipStream_t) stream>>>((T *)input, (T *)scale, (T *)bias, (T *)output, eps, behindsize);
    }
    else if (behindsize > 15)
    {
        int BLOCK_DIM_x = 16;
        int BLOCK_DIM_y = 64;
        int num_block_x = (num_blocks + BLOCK_DIM_y - 1) / BLOCK_DIM_y;
        dim3 block_dim(BLOCK_DIM_x, BLOCK_DIM_y, 1);
        dim3 grid_dim(num_block_x, 1, 1);

        warpLayernormKernel<T, 16, 64>
            <<<grid_dim, block_dim, 0, (hipStream_t) stream>>>((T *)input, (T *)scale, (T *)bias, (T *)output, eps, behindsize);
    }
    else if (behindsize > 7)
    {
        int BLOCK_DIM_x = 8;
        int BLOCK_DIM_y = 128;
        int num_block_x = (num_blocks + BLOCK_DIM_y - 1) / BLOCK_DIM_y;
        dim3 block_dim(BLOCK_DIM_x, BLOCK_DIM_y, 1);
        dim3 grid_dim(num_block_x, 1, 1);

        warpLayernormKernel<T, 8, 128>
            <<<grid_dim, block_dim, 0, (hipStream_t) stream>>>((T *)input, (T *)scale, (T *)bias, (T *)output, eps, behindsize);
    }
    else
    {
        int BLOCK_DIM_x = 4;
        int BLOCK_DIM_y = 256;
        int num_block_x = (num_blocks + BLOCK_DIM_y - 1) / BLOCK_DIM_y;
        dim3 block_dim(BLOCK_DIM_x, BLOCK_DIM_y, 1);
        dim3 grid_dim(num_block_x, 1, 1);

        warpLayernormKernel<T, 4, 256>
            <<<grid_dim, block_dim, 0, (hipStream_t) stream>>>((T *)input, (T *)scale, (T *)bias, (T *)output, eps, behindsize);
    }

}

infiniopStatus_t cudaLayerNorm(LayerNormCudaDescriptor_t desc,                    
                             void const *x, void const *w, void const *b, void *y,
                             void *stream) {
    if (hipSetDevice(desc->device_id) != hipSuccess) {
        return STATUS_BAD_DEVICE;
    }
    if (dtype_eq(desc->dtype, F16)) {
        layer_norm_nv_gpu<half>(desc, x, w, b, y, stream);
        return STATUS_SUCCESS;
    }
    if (dtype_eq(desc->dtype, F32)) {
        layer_norm_nv_gpu<float>(desc, x, w, b, y, stream);
        return STATUS_SUCCESS;
    }
    return STATUS_BAD_TENSOR_DTYPE;
}
