#include "hip/hip_runtime.h"
#include "../../../devices/cuda/handle_pool.h"
#include "../../utils.h"
#include "../blas.h"
#include "matmul_cuda.h"
#include <hipblas.h>
#include <hip/hip_fp16.h>

MatmulCudaDescriptor::MatmulCudaDescriptor(Device device) {
    this->device = device;
    get_cublas_pool();
}

void matmul_nv_gpu_f16(Tensor c, float beta, Tensor a, Tensor b, float alpha, void *stream) {
    auto info = MatmulInfo(c, a, b);

    auto alpha_f16 = __float2half(alpha);
    auto beta_f16 = __float2half(beta);

    auto op_a = info.a_matrix.row_stride == 1 ? HIPBLAS_OP_N : HIPBLAS_OP_T;
    auto op_b = info.b_matrix.row_stride == 1 ? HIPBLAS_OP_N : HIPBLAS_OP_T;

    use_cublas((hipStream_t) stream,
               [&](hipblasHandle_t handle) { hipblasGemmStridedBatchedEx(
                                                handle,
                                                op_a,
                                                op_b,
                                                info.m,
                                                info.n,
                                                info.k,
                                                &alpha_f16,
                                                info.a_ptr,
                                                HIP_R_16F,
                                                info.a_matrix.ld(),
                                                info.a_matrix.stride,
                                                info.b_ptr,
                                                HIP_R_16F,
                                                info.b_matrix.ld(),
                                                info.b_matrix.stride,
                                                &beta_f16,
                                                info.c_ptr,
                                                HIP_R_16F,
                                                info.c_matrix.ld(),
                                                info.c_matrix.stride,
                                                info.batch,
                                                HIPBLAS_COMPUTE_16F,
                                                CUBLAS_GEMM_DEFAULT_TENSOR_OP); });
}
